#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blocksize 128

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void upSweep(int n, int pow2dPlus1, int pow2d, int *odata, bool reachedRoot)
		{
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) {
				return;
			}

			if (reachedRoot) {
				odata[n - 1] = 0;
			}
			else {
				index *= pow2dPlus1;
				if (index < n)
					odata[index + pow2dPlus1 - 1] += odata[index + pow2d - 1];
			}
		}

		__global__ void downSweep(int n, int pow2dPlus1, int pow2d, int *odata)
		{
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) {
				return;
			}

			index *= pow2dPlus1;
			if (index < n) {
				int t = odata[index + pow2d - 1];
				odata[index + pow2d - 1] = odata[index + pow2dPlus1 - 1];
				odata[index + pow2dPlus1 - 1] += t;
			}
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			dim3 fullBlocksPerGrid((n + blocksize - 1) / blocksize);
			
			// Get the next power of 2
			int currPow = ilog2ceil(n) - 1;
			int nextPow = 2 << currPow;

			int *temp = new int[nextPow];
			for (int i = 0; i < nextPow; i++) {
				if (i < n) {
					temp[i] = idata[i];
				}
				// Fill the rest of the array with 0 if not a power of 2.
				else {
					temp[i] = 0;
				}
			}

			int *out;
			hipMalloc((void**)&out, nextPow * sizeof(int));
			checkCUDAError("hipMalloc out failed!");
			hipMemcpy(out, temp, sizeof(int) * nextPow, hipMemcpyHostToDevice);

			timer().startGpuTimer();
            // TODO

			// Up-Sweep
			for (int d = 0; d <= ilog2ceil(nextPow) - 1; d++) {
				int pow2dPlus1 = pow(2, d + 1);
				int pow2d = pow(2, d);

				// If we hit the end of the depth then we should be writing to the very last spot in the array.
				bool reachedRoot = (d == ilog2ceil(nextPow) - 1);
				upSweep << < fullBlocksPerGrid, blocksize >> > (nextPow, pow2dPlus1, pow2d, out, reachedRoot);
			}

			// Down-Sweep
			for (int d = ilog2ceil(nextPow) - 1; d >= 0; d--) {
				int pow2dPlus1 = pow(2, d + 1);
				int pow2d = pow(2, d);
			
				downSweep << < fullBlocksPerGrid, blocksize >> > (nextPow, pow2dPlus1, pow2d, out);
			}

            timer().endGpuTimer();

			// Copy final values into odata
			hipMemcpy(odata, out, sizeof(int) * nextPow, hipMemcpyDeviceToHost);

			delete[]temp;
			hipFree(out);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
		int compact(int n, int *odata, const int *idata) {
			dim3 fullBlocksPerGrid((n + blocksize - 1) / blocksize);

			int *dev_in;
			hipMalloc((void**)&dev_in, sizeof(int) * n);
			hipMemcpy(dev_in, idata, sizeof(int) * n, hipMemcpyHostToDevice);

			int *temp;
			hipMalloc((void**)&temp, sizeof(int) * n);

			timer().startGpuTimer();
			// TODO

			int *dev_bools;
			hipMalloc((void**)&dev_bools, sizeof(int) * n);
			//int *bools = new int[n];
			StreamCompaction::Common::kernMapToBoolean << < fullBlocksPerGrid, blocksize >> > (n, dev_bools, dev_in);
			//hipMemcpy(bools, dev_bools, sizeof(int) * n, hipMemcpyDeviceToHost);

			/*hipMemcpy(odata, dev_bools, sizeof(int) * n, hipMemcpyDeviceToHost);
			exclusiveScan(n, temp, odata);

			for (int i = 0; i < n; i++) {
				printf("INDICES: %i\n", temp[i]);
			}*/

			/*int *indices;
			hipMalloc((void**)&indices, sizeof(int) * n);
			hipMemcpy()*/







			/*StreamCompaction::Common::kernScatter << < fullBlocksPerGrid, blocksize >> > (n, dev_out, dev_in, dev_bools, dev_indices);
			hipMemcpy(odata, dev_out, sizeof(int) * n, hipMemcpyDeviceToHost);*/

			timer().endGpuTimer();
			return -1;
		}
    }
}
