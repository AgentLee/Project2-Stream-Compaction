#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blocksize 128

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void upSweep(int n, int pow2dPlus1, int pow2d, int *odata)
		{
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) {
				return;
			}

			index *= pow2dPlus1;
			if (index < n)
				odata[index + pow2dPlus1 - 1] += odata[index + pow2d - 1];
		}

		__global__ void downSweep(int n, int pow2dPlus1, int pow2d, int *odata)
		{
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) {
				return;
			}

			index *= pow2dPlus1;
			if (index < n) {
				int t = odata[index + pow2d - 1];
				odata[index + pow2d - 1] = odata[index + pow2dPlus1 - 1];
				odata[index + pow2dPlus1 - 1] += t;
			}
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			dim3 fullBlocksPerGrid((n + blocksize - 1) / blocksize);
			
			int *out;
			hipMalloc((void**)&out, n * sizeof(int));
			hipMemcpy(out, idata, sizeof(int) * n, hipMemcpyHostToDevice);

			timer().startGpuTimer();
            // TODO

			// Get the next power of 2
			int currPow = ilog2ceil(n) - 1;
			int nextPow = 2 << currPow;

			// Up-Sweep
			for (int d = 0; d <= ilog2ceil(n) - 1; d++) {
				int pow2dPlus1 = pow(2, d + 1);
				int pow2d = pow(2, d);

				upSweep << < fullBlocksPerGrid, blocksize >> > (n, pow2dPlus1, pow2d, out);
			}

			// Gotta find a better solution to this. lol
			hipMemcpy(odata, out, sizeof(int) * n, hipMemcpyDeviceToHost);
			odata[n - 1] = 0;
			hipMemcpy(out, odata, sizeof(int) * n, hipMemcpyHostToDevice);

			// Down-Sweep
			for (int d = ilog2ceil(n) - 1; d >= 0; d--) {
				int pow2dPlus1 = pow(2, d + 1);
				int pow2d = pow(2, d);
			
				downSweep << < fullBlocksPerGrid, blocksize >> > (n, pow2dPlus1, pow2d, out);
			}

			hipMemcpy(odata, out, sizeof(int) * n, hipMemcpyDeviceToHost);

			hipFree(out);

            timer().endGpuTimer();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}
